#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2018)
 * Projekt c.2 (CUDA)
 * Login: xpawlu00
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>
#include <hip/hip_vector_types.h>
#include "nbody.h"

int main(int argc, char **argv)
{
    FILE *fp;
    struct timeval t1, t2;
    int N;
    float dt;
    int steps;
    int thr_blc;

    // parametry
    if (argc != 7)
    {
        printf("Usage: nbody <N> <dt> <steps> <thr/blc> <input> <output>\n");
        exit(1);
    }
    N = atoi(argv[1]);
    dt = atof(argv[2]);
    steps = atoi(argv[3]);
    thr_blc = atoi(argv[4]);

    printf("N: %d\n", N);
    printf("dt: %f\n", dt);
    printf("steps: %d\n", steps);
    printf("threads/block: %d\n", thr_blc);

    // alokace pameti na CPU
    t_particles particles_cpu;
    hipHostAlloc(&particles_cpu.pos, N * sizeof(float4), hipHostMallocDefault);
    hipHostAlloc(&particles_cpu.vel, N * sizeof(float4), hipHostMallocDefault);
    // nacteni castic ze souboru
    fp = fopen(argv[5], "r");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[5]);
        exit(1);
    }
    particles_read(fp, particles_cpu, N);
    fclose(fp);
    t_particles particles_gpu[2];
    for (int i = 0; i < 2; i++) {
        hipMalloc(&particles_gpu[i].pos, N * sizeof(float4));
        hipMalloc(&particles_gpu[i].vel, N * sizeof(float4));
        hipMemcpy(particles_gpu[i].pos, particles_cpu.pos, N * sizeof(float4), hipMemcpyHostToDevice);
        hipMemcpy(particles_gpu[i].vel, particles_cpu.vel, N * sizeof(float4), hipMemcpyHostToDevice);
    }
    // vypocet velikosti vypocetniho gridu
    size_t grid = N / thr_blc + (N % thr_blc == 0 ? 0 : 1);
    printf("grid size is %d\n", grid);
    
    // vypocet
    gettimeofday(&t1, 0);

    t_particles *p_in;
    t_particles *p_out;
    for (int s = 0; s < steps; ++s)
    {
        p_in = &particles_gpu[s % 2];
        p_out = &particles_gpu[(s + 1) % 2];
        calculate_velocity<<<grid, thr_blc, thr_blc * 2 * sizeof(float4)>>>(*p_in, *p_out, N, dt);
    }
    hipDeviceSynchronize();
    gettimeofday(&t2, 0);

    // cas
    double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Time: %f s\n", t);
    // kopirovani castic zpatky na CPU
    hipMemcpy(particles_cpu.pos, p_out->pos, N * sizeof(float4), hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.vel, p_out->vel, N * sizeof(float4), hipMemcpyDeviceToHost);
    // ulozeni castic do souboru
    fp = fopen(argv[6], "w");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[6]);
        exit(1);
    }
    particles_write(fp, particles_cpu, N);
    fclose(fp);
    for (int i = 0; i < 2; i++) {
        hipFree(particles_gpu[i].pos);
        hipFree(particles_gpu[i].vel);
    }
    hipHostFree(particles_cpu.pos);   
    hipHostFree(particles_cpu.vel);
    return 0;
}
