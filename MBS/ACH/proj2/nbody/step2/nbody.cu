#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2018)
 * Projekt c.2 (CUDA)
 * Login: xpawlu00
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

__global__ void calculate_velocity(t_particles p_in, t_particles p_out, int N, float dt) {
  int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  extern __shared__ float4 shM[];
  // promenna vel, do ktere se budou akumulovat vypocty
  float4 vel = {0.0f, 0.0f, 0.0f, 0.0f};
  float4* shM_pos = shM;
  float4* shM_vel = shM + blockDim.x;
  float r;
  float4 d;
  float4 p1_pos, p1_vel;
  if (idx < N) {
    p1_pos = p_in.pos[idx];
    p1_vel = p_in.vel[idx];
  }  
  // pocet "dlazdic" == pocet bloku
  for (int tile = 0; tile < gridDim.x; ++tile) {
    int blockTile = blockDim.x * tile;
    int index = blockTile + threadIdx.x;
    // nacteni "dlazdice" do sdilene pameti - musi zde pracovat i vlakna, jejichz idx > N!
    if (index < N) {
      shM_pos[threadIdx.x] = p_in.pos[index];
      shM_vel[threadIdx.x] = p_in.vel[index];
    }    
    __syncthreads();
    if (idx < N) {
      int limit = (tile + 1) * blockDim.x > N ? (N - blockTile) : blockDim.x;
      for (int i = 0; i < limit; i++) {
        // stejny prvek
        if (idx == blockTile + i)
          continue;
        float4 p2_pos = shM_pos[i];
        float4 p2_vel = shM_vel[i];
        d.x = p2_pos.x - p1_pos.x;
        d.y = p2_pos.y - p1_pos.y;
        d.z = p2_pos.z - p1_pos.z;
        r = sqrt(d.x*d.x + d.y*d.y + d.z*d.z);
        if (r > 0.0f && r < COLLISION_DISTANCE) {
          float weight = p1_vel.w / p2_vel.w;
          vel.x += (((weight - 1) * p1_vel.x + 2 * p2_vel.x)/(1 + weight) - p1_vel.x);
          vel.y += (((weight - 1) * p1_vel.y + 2 * p2_vel.y)/(1 + weight) - p1_vel.y);
          vel.z += (((weight - 1) * p1_vel.z + 2 * p2_vel.z)/(1 + weight) - p1_vel.z);
        }
        if (r > COLLISION_DISTANCE) {
          vel.x += (G * p2_vel.w) / pow(r, 3) * (d.x) * dt;
          vel.y += (G * p2_vel.w) / pow(r, 3) * (d.y) * dt;
          vel.z += (G * p2_vel.w) / pow(r, 3) * (d.z) * dt;
        }
      }
    }    
    __syncthreads();
  }
  if (idx < N) {
    float4 tmp_vel = {p1_vel.x + vel.x, p1_vel.y + vel.y, p1_vel.z + vel.z, p1_vel.w};
    p_out.vel[idx] = tmp_vel;
    p_out.pos[idx].x = p1_pos.x + tmp_vel.x * dt;
    p_out.pos[idx].y = p1_pos.y + tmp_vel.y * dt;
    p_out.pos[idx].z = p1_pos.z + tmp_vel.z * dt;
  }
}

__global__ void calculate_gravitation_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  if (idx >= N)
    return;
  float r;
  float4 d;
  float4 p1_pos = p.pos[idx];
  float4 vel = {0.0f, 0.0f, 0.0f};
  for (int i = 0; i < N; i++) { 
    if (i == idx)
      continue;
    float4 p2_pos = p.pos[i];
    d.x = p2_pos.x - p1_pos.x;
    d.y = p2_pos.y - p1_pos.y;
    d.z = p2_pos.z - p1_pos.z;
    r = sqrt(d.x*d.x + d.y*d.y + d.z*d.z);
    if (r > COLLISION_DISTANCE) {
      float p2_weight = p.vel[i].w;
      vel.x += (G * p2_weight) / pow(r, 3) * (d.x) * dt;
      vel.y += (G * p2_weight) / pow(r, 3) * (d.y) * dt;
      vel.z += (G * p2_weight) / pow(r, 3) * (d.z) * dt;
    }
  }
  tmp_vel.vel[idx] = vel;
}

__global__ void calculate_collision_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  if (idx >= N)
    return;
  float4 d;
  float r;
  float4 p1_pos = p.pos[idx];
  float4 tmp_vel_local = tmp_vel.vel[idx];
  float4 p1_vel = p.vel[idx];
  float weight1 = p1_vel.w;
  for (int i = 0; i < N; i++) {  
    if (i == idx)
      continue;
    float4 p2_pos = p.pos[i];
    d.x = p2_pos.x - p1_pos.x;
    d.y = p2_pos.y - p1_pos.y;
    d.z = p2_pos.z - p1_pos.z;
    float4 p2_vel = p.vel[i];
    float weight2 = p2_vel.w;
    r = sqrt(d.x*d.x + d.y*d.y + d.z*d.z);
    if (r > 0.0f && r < COLLISION_DISTANCE) {
      float weight = weight1 / weight2;
      tmp_vel_local.x += (((weight - 1) * p1_vel.x + 2 * p2_vel.x)/(1 + weight) - p1_vel.x);
      tmp_vel_local.y += (((weight - 1) * p1_vel.y + 2 * p2_vel.y)/(1 + weight) - p1_vel.y);
      tmp_vel_local.z += (((weight - 1) * p1_vel.z + 2 * p2_vel.z)/(1 + weight) - p1_vel.z);
    }
  }
  tmp_vel.vel[idx] = tmp_vel_local;
}

__global__ void update_particle(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  if (idx >= N)
    return;
  p.vel[idx].x += tmp_vel.vel[idx].x;
  p.vel[idx].y += tmp_vel.vel[idx].y;
  p.vel[idx].z += tmp_vel.vel[idx].z;
  p.pos[idx].x += p.vel[idx].x * dt;
  p.pos[idx].y += p.vel[idx].y * dt;
  p.pos[idx].z += p.vel[idx].z * dt;

}

__host__ void particles_read(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
        fscanf(fp, "%f %f %f %f %f %f %f \n",
    &p.pos[i].x, &p.pos[i].y, &p.pos[i].z,
    &p.vel[i].x, &p.vel[i].y, &p.vel[i].z, &p.vel[i].w);
    }
}

__host__  void particles_write(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
        fprintf(fp, "%f %f %f %f %f %f %f \n",
    p.pos[i].x, p.pos[i].y, p.pos[i].z,
    p.vel[i].x, p.vel[i].y, p.vel[i].z, p.vel[i].w);
    }
}
