#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2018)
 * Projekt c.2 (CUDA)
 * Login: xpawlu00
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>
#include <hip/hip_vector_types.h>
#include "nbody.h"

int main(int argc, char **argv)
{
    FILE *fp;
    struct timeval t1, t2;
    int N;
    float dt;
    int steps;
    int thr_blc;

    // parametry
    if (argc != 7)
    {
        printf("Usage: nbody <N> <dt> <steps> <thr/blc> <input> <output>\n");
        exit(1);
    }
    N = atoi(argv[1]);
    dt = atof(argv[2]);
    steps = atoi(argv[3]);
    thr_blc = atoi(argv[4]);

    printf("N: %d\n", N);
    printf("dt: %f\n", dt);
    printf("steps: %d\n", steps);
    printf("threads/block: %d\n", thr_blc);
    // alokace pameti na CPU
    t_particles particles_cpu;
    hipHostAlloc(&particles_cpu.pos, N * sizeof(float4), hipHostMallocDefault);
    hipHostAlloc(&particles_cpu.vel, N * sizeof(float4), hipHostMallocDefault);
    // nacteni castic ze souboru
    fp = fopen(argv[5], "r");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[5]);
        exit(1);
    }
    particles_read(fp, particles_cpu, N);
    fclose(fp);
    t_particles particles_gpu;
    t_velocities tmp_velocities_gpu;
    // alokace pameti na GPU
    hipMalloc(&particles_gpu.pos, N * sizeof(float4));
    hipMalloc(&particles_gpu.vel, N * sizeof(float4));
    // kopie dat z cpu na gpu
    hipMemcpy(particles_gpu.pos, particles_cpu.pos, N * sizeof(float4), hipMemcpyHostToDevice);
    hipMemcpy(particles_gpu.vel, particles_cpu.vel, N * sizeof(float4), hipMemcpyHostToDevice);
    // alokace pomocne struktury na gpu
    hipMalloc(&tmp_velocities_gpu.vel, N * sizeof(float4));
    // vypocet gridu
    size_t grid = N / thr_blc + (N % thr_blc == 0 ? 0 : 1);    
    // vypocet
    gettimeofday(&t1, 0);
    for (int s = 0; s < steps; ++s)
    {
        calculate_gravitation_velocity<<<grid, thr_blc>>>(particles_gpu,tmp_velocities_gpu, N, dt);
        calculate_collision_velocity<<<grid, thr_blc>>>(particles_gpu,tmp_velocities_gpu, N, dt);
        update_particle<<<grid, thr_blc>>>(particles_gpu,tmp_velocities_gpu, N, dt);
    }
    hipDeviceSynchronize();
    gettimeofday(&t2, 0);
    // cas
    double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Time: %f s\n", t);
    // kopirovani castic zpatky na CPU */
    hipMemcpy(particles_cpu.pos, particles_gpu.pos, N * sizeof(float4), hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.vel, particles_gpu.vel, N * sizeof(float4), hipMemcpyDeviceToHost);
    // ulozeni castic do souboru
    fp = fopen(argv[6], "w");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[6]);
        exit(1);
    }
    particles_write(fp, particles_cpu, N);
    fclose(fp);
    hipHostFree(particles_cpu.pos);   
    hipHostFree(particles_cpu.vel);
    hipFree(particles_gpu.pos);
    hipFree(particles_gpu.vel);
    hipFree(tmp_velocities_gpu.vel);
    return 0;
}