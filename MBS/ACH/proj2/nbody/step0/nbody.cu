#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2018)
 * Projekt c.2 (CUDA)
 * Login: xpawlu00
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

__global__ void calculate_gravitation_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  if (idx >= N)
    return;
  float r;
  float4 d;
  float4 p1_pos = p.pos[idx];
  float4 vel = {0.0f, 0.0f, 0.0f};
  for (int i = 0; i < N; i++) { 
    if (i == idx)
      continue;
    float4 p2_pos = p.pos[i];
    d.x = p2_pos.x - p1_pos.x;
    d.y = p2_pos.y - p1_pos.y;
    d.z = p2_pos.z - p1_pos.z;
    r = sqrt(d.x*d.x + d.y*d.y + d.z*d.z);
    if (r > COLLISION_DISTANCE) {
      float p2_weight = p.vel[i].w;
      vel.x += (G * p2_weight) / pow(r, 3) * (d.x) * dt;
      vel.y += (G * p2_weight) / pow(r, 3) * (d.y) * dt;
      vel.z += (G * p2_weight) / pow(r, 3) * (d.z) * dt;
    }
  }
  tmp_vel.vel[idx] = vel;
}

__global__ void calculate_collision_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  if (idx >= N)
    return;
  float4 d;
  float r;
  float4 p1_pos = p.pos[idx];
  float4 tmp_vel_local = tmp_vel.vel[idx];
  float4 p1_vel = p.vel[idx];
  float weight1 = p1_vel.w;
  for (int i = 0; i < N; i++) {  
    if (i == idx)
      continue;
    float4 p2_pos = p.pos[i];
    d.x = p2_pos.x - p1_pos.x;
    d.y = p2_pos.y - p1_pos.y;
    d.z = p2_pos.z - p1_pos.z;
    float4 p2_vel = p.vel[i];
    float weight2 = p2_vel.w;
    r = sqrt(d.x*d.x + d.y*d.y + d.z*d.z);
    if (r > 0.0f && r < COLLISION_DISTANCE) {
      float weight = weight1 / weight2;
      tmp_vel_local.x += (((weight - 1) * p1_vel.x + 2 * p2_vel.x)/(1 + weight) - p1_vel.x);
      tmp_vel_local.y += (((weight - 1) * p1_vel.y + 2 * p2_vel.y)/(1 + weight) - p1_vel.y);
      tmp_vel_local.z += (((weight - 1) * p1_vel.z + 2 * p2_vel.z)/(1 + weight) - p1_vel.z);
    }
  }
  tmp_vel.vel[idx] = tmp_vel_local;
}

__global__ void update_particle(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  if (idx >= N)
    return;
  p.vel[idx].x += tmp_vel.vel[idx].x;
  p.vel[idx].y += tmp_vel.vel[idx].y;
  p.vel[idx].z += tmp_vel.vel[idx].z;
  p.pos[idx].x += p.vel[idx].x * dt;
  p.pos[idx].y += p.vel[idx].y * dt;
  p.pos[idx].z += p.vel[idx].z * dt;

}

__host__ void particles_read(FILE *fp, t_particles &p, int N)
{
  for (int i = 0; i < N; i++) {
    fscanf(fp, "%f %f %f %f %f %f %f \n",
    &p.pos[i].x, &p.pos[i].y, &p.pos[i].z,
    &p.vel[i].x, &p.vel[i].y, &p.vel[i].z, &p.vel[i].w);
  }
}

__host__  void particles_write(FILE *fp, t_particles &p, int N)
{
  for (int i = 0; i < N; i++) {
    fprintf(fp, "%f %f %f %f %f %f %f \n",
    p.pos[i].x, p.pos[i].y, p.pos[i].z,
    p.vel[i].x, p.vel[i].y, p.vel[i].z, p.vel[i].w);
  }
}
